#include "hip/hip_runtime.h"
#include "cuda_grid_functions.h"

__global__ void countNeighbours(int *grid, int *resultGrid, int columns, int rows) {
  
	/* ignore the wrapper threads */
	int x = blockDim.x*blockIdx.x + threadIdx.x;

	/*  check if this thread is part of the right wrapper column */
	int rcol = (columns * rows - x ) % columns;
	
	if( ( x < (columns * rows)) && 
	    ( rcol !=1 ) 		    && 
	    ( x % (columns) != 0)   &&
	    ( x > columns )         &&
	    ( x < ((columns * rows - columns) -1))
	   ) {
		int sum = grid[x + 1]       + grid[x - 1] + 
	    		  grid[x + columns] + grid[x + columns + 1] + grid[x + columns - 1] +
	    		  grid[x - columns] + grid[x - columns + 1] + grid[x - columns - 1];

	    if (sum <= 1) {
	    	resultGrid[x] = DEAD;
	    } else if ((sum == 2 || sum == 3) && (grid[x] == ALIVE)) {
	        resultGrid[x] = ALIVE;
	    } else if ((sum >=4) && (sum <= 8)) {
	        resultGrid[x] = DEAD;
	    } else if ((sum == 3) && (grid[x] == DEAD)) {
			resultGrid[x] = ALIVE;
	    } else if ((sum == 2) && (grid[x] == DEAD)) {
	        resultGrid[x] = DEAD;
	    }
	    
	}
    __syncthreads();
}

int main(int argc, char **argv) {

	int *host_grid          = NULL;
	int *previous_host_grid = NULL;
	int *host_grid_result   = NULL;
	int *device_grid        = NULL;
	int *device_grid_result = NULL;

	int repetitions;
	bool check_termination = false;
	bool print             = false;
	bool readfile          = false;
	int grid_size;

	int err = parse_command_line(argc, argv, &host_grid, &host_grid_result, 
														 &previous_host_grid, 
														 &repetitions, 
														 &grid_size, 
														 &check_termination, 
														 &print, &readfile);

	if (err < 0) {
		exit(0);
	}

	printf("grid size: %d\n repetitions: %d\n", grid_size, repetitions);

	copy_wrapper(host_grid, grid_size);

	int number_of_blocks  = (grid_size * grid_size) / THREADS_PER_BLOCK;
	int remaining_threads = (grid_size * grid_size) % THREADS_PER_BLOCK;

	if (remaining_threads > 0) {
		number_of_blocks++;
	}

	printf("Number of blocks: %d\n", number_of_blocks );

	hipDeviceSynchronize();

	hipMalloc((void **)&device_grid, 	     WRAPPER_SIZE * sizeof(int));
	hipMalloc((void **)&device_grid_result, WRAPPER_SIZE * sizeof(int));
	hipMemset(device_grid_result, 0,        WRAPPER_SIZE * sizeof(int));
	hipMemcpy(device_grid, host_grid, 	     WRAPPER_SIZE * sizeof(int), hipMemcpyHostToDevice);

	if (print) {
		print_grid(host_grid, grid_size);
		printf("~~~~");
	}

	clock_t start = clock();

 	for(int rep = 0; rep < repetitions; rep++) {
		countNeighbours<<<number_of_blocks, THREADS_PER_BLOCK>>>(device_grid, device_grid_result, grid_size + 2, grid_size + 2);
		
		hipDeviceSynchronize();
		hipMemcpy(host_grid_result, device_grid_result, WRAPPER_SIZE * sizeof(int), hipMemcpyDeviceToHost);

		if (print) {
			print_grid(host_grid_result, grid_size);
			printf("~~~~");
		}

		/* If we want to check for termination we do it every 10 reps. */
		if (check_termination) {
			if (rep % 10 == 0) {
				if (equalGrids(host_grid_result, host_grid, grid_size) || empty_grid(host_grid_result, grid_size)) {
				printf("Grid unchanged or empty. Exiting...\n");
					break;
				}
			}
		}

		copy_wrapper(host_grid_result, grid_size);
		
		/* Copy again the new grid back to device. */
		hipMemcpy(device_grid, host_grid_result, WRAPPER_SIZE * sizeof(int), hipMemcpyHostToDevice);	

		/* Swap for future checks. */
		int *swap;
        swap             = host_grid_result;
        host_grid_result = host_grid;
        host_grid        = swap;
	}

	clock_t end = clock();
	float seconds = (float)(end - start) / CLOCKS_PER_SEC;

    printf("Execution time: %.6f secs. \n", seconds);

    free(host_grid_result);
    free(host_grid);
    hipFree(device_grid);
    hipFree(device_grid_result);

	exit(EXIT_SUCCESS);
}